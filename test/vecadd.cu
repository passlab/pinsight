#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>

// Vector addition kernel
__global__ void VecAdd(const int *A, const int *B, int *C, int N) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < N)
        C[i] = A[i] + B[i];
}

// Initialize a vector
static void initVec(int *vec, int n) {
    for (int i = 0; i < n; i++)
        vec[i] = i;
}

int main(int argc, char *argv[]) {
    hipCtx_t context = 0;
    hipDevice_t device = 0;
    int N = 50000000;
    size_t size = N * sizeof(int);
    int threadsPerBlock = 0;
    int blocksPerGrid = 0;
    int sum, i;
    int *h_A, *h_B, *h_C;
    int *d_A, *d_B, *d_C;

    hipInit(0);
    hipCtxCreate(&context, 0, device);

    // Allocate input vectors h_A and h_B in host memory
    h_A = (int *) malloc(size);
    h_B = (int *) malloc(size);
    h_C = (int *) malloc(size);

    // Initialize input vectors
    initVec(h_A, N);
    initVec(h_B, N);
    memset(h_C, 0, size);

    // Allocate vectors in device memory
    hipMalloc((void **) &d_A, size);
    hipMalloc((void **) &d_B, size);
    hipMalloc((void **) &d_C, size);

    // Copy vectors from host memory to device memory
    hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

    // Invoke kernel
    threadsPerBlock = 256;
    blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;

    VecAdd << < blocksPerGrid, threadsPerBlock >> > (d_A, d_B, d_C, N);
    hipDeviceSynchronize();

    // Copy result from device memory to host memory
    // h_C contains the result in host memory
    hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);
    hipDeviceSynchronize();

    // Verify result
    for (i = 0; i < N; ++i) {
        sum = h_A[i] + h_B[i];
        if (h_C[i] != sum) {
            printf("kernel execution FAILED: %d vs %d\n", h_C[i], sum);
            break;
        }
    }

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    free(h_A);
    free(h_B);
    free(h_C);

    hipDeviceSynchronize();
    return 0;
}
